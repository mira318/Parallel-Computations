
#include <hip/hip_runtime.h>
#include <cstdio>
#include <functional>
#include <iostream>
#include <random>

#define BLOCKSIZE 256


void FillMatrix(float* matrix, int height, int width) {
    std::mt19937 gen(time(0));
    std::uniform_real_distribution<float> distribution(-1.0f, 1.0f);
    auto generate = std::bind(distribution, gen);
    for (int i = 0; i < height * width; ++i) {
        matrix[i] = generate();
    }
}

void PrintMatrix(float *matrix, int height, int width) {

    for (int i = 0; i < height; ++i) {
        for (int j = 0; j < width; ++j) {
            std::cout << matrix[i * width + j] << " ";
        }
	std::cout << std::endl;
    }
}


__global__
void kernel_mul(float *A, float *B, float *C, int mid_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int height = blockDim.x * gridDim.x;
    int width = blockDim.y * gridDim.y;

    C[i * width + j] = 0.0f;
    for (int k = 0; k < mid_size; ++k) {
        C[i * width + j] += A[i * mid_size + k] * B[k * width + j];
    }
}

__global__
void kernel_my_mul(float *A, float *B, float *C, int mid_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int height = blockDim.x * gridDim.x;
    int width = blockDim.y * gridDim.y;

    __shared__ float block_a[BLOCKSIZE];
    __shared__ float block_b[BLOCKSIZE];

    block_a[threadIdx.y] = A[i * width + threadIdx.y];
    block_b[threadIdx.x] = B[threadIdx.x * height + j];

    __syncthreads();

    C[i * width + j] = 0.0f;
    for (int k = 0; k < mid_size; ++k) {
	C[i * width + j] += block_a[k] * block_b[k];
    }
}

void try_both_multiplications(float *h_A, float *h_B, float *h_C) {
    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, sizeof(float) * 128 * 384);
    hipMalloc(&d_B, sizeof(float) * 384 * 256);
    hipMalloc(&d_C, sizeof(float) * 128 * 256);

    hipMemcpy(d_A, h_A, sizeof(float) * 128 * 384, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * 384 * 256, hipMemcpyHostToDevice);

    // kernel call
    dim3 num_blocks(8, 16);
    dim3 block_size(16, 16);

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    kernel_mul<<<num_blocks, block_size>>>(d_A, d_B, d_C, 384);

    hipEventRecord(stop);
    hipMemcpy(h_C, d_C, sizeof(float) * 128 * 256, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop);

    float milliseconds = 0;

    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "banal elapsed in " << milliseconds << std::endl;

    PrintMatrix(h_C, 128, 256);

    hipEvent_t start2;
    hipEvent_t stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);

    hipEventRecord(start2);
    kernel_my_mul<<<num_blocks, block_size>>>(d_A, d_B, d_C, 384);

    hipEventRecord(stop2);
    hipMemcpy(h_C, d_C, sizeof(float) * 128 * 256, hipMemcpyDeviceToHost);
    hipEventSynchronize(stop2);

    float milliseconds2 = 0;

    hipEventElapsedTime(&milliseconds2, start2, stop2);
    std::cout << "shared memory elapsed in " << milliseconds2 << std::endl;

    PrintMatrix(h_C, 128, 256);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main() {

    float *h_A;
    float *h_B;
    float *h_C;

    // h_A 128 * 384
    // h_B 384 * 256
    // h_C 128 * 256

    h_A = new float[128 * 384];
    h_B = new float[384 * 256];
    h_C = new float[128 * 256];

    FillMatrix(h_A, 128, 384);
    FillMatrix(h_B, 384, 256);
    try_both_multiplications(h_A, h_B, h_C);

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    return 0;
}
